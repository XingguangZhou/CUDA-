#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

__global__ void AddArray(int *a, int *b, int *c)
{
	int i = threadIdx.x;
	printf("GPU!\n");
	c[i] = a[i] + b[i];
}


int main(void)
{
	int h_a[4] = { 0, 0, 0, 1 };
	int h_b[4] = { 1, 2, 3, 4 };
	int h_c[4] = { 0 };
	int *d_a = NULL;
	int *d_b = NULL;
	int *d_c = NULL;
	int i;

	dim3 block(4);
	dim3 grid((4 + block.x - 1) / block.x);

	hipMalloc((int **)&d_a, sizeof(int)* 4);
	hipMalloc((int **)&d_b, sizeof(int)* 4);
	hipMalloc((int **)&d_c, sizeof(int)* 4);

	hipMemcpy(d_a, h_a, sizeof(int)* 4, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int)* 4, hipMemcpyHostToDevice);

	AddArray << <grid, block >> >(d_a, d_b, d_c);
	//hipDeviceSynchronize(); // 强制设备与主机相互同步，要求核函数运算完成后，CPU再进行计算，可以试验，如果将此语句去掉后查看结果。
	for (i = 0; i < 3; i++)
	{
		printf("CPU!\n");
	}

	hipMemcpy(h_c, d_c, sizeof(int)* 4, hipMemcpyDeviceToHost);

	for (i = 0; i < 4; i++)
	{
		printf("%d\t", h_c[i]);
	}
	printf("\n");

	// free the memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipDeviceReset();
	return 0;
}
