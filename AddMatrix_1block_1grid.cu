#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

// initialize the matrix
void initalMatrix(int *matrix, int nx, int ny)
{
	int i, j;
	int cnt = 0;
	for (j = 0; j < ny; j++)
	{
		for (i = 0; i < nx; i++)
		{
			matrix[cnt] = cnt;
			cnt++;
		}
	}
}

__global__ void AddMatrixOnGPU(int *A, int *B, int *C, int nx, int ny)
{
	int x, y;
	int idx;
	x = threadIdx.x + blockIdx.x * blockDim.x;
	y = threadIdx.y + blockIdx.y * blockDim.y;
	idx = y*nx + nx;
	if (x < nx && y < ny)
	{
		C[idx] = A[idx] + B[idx];
	}
}

__global__ void Book(int *A, int *B, int *C, int nx, int ny)
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;
	if (ix < nx)
	{
		for (int iy = 0; iy < ny; iy++)
		{
			int idx = iy*nx + ix;
			C[idx] = A[idx] + B[idx];
		}
	}
	
}

void AddMatrixOnCPU(int *A, int *B, int *C, int nx, int ny)
{
	int i, j;
	int cnt = 0;
	for (j = 0; j < ny; j++)
	{
		for (i = 0; i < nx; i++)
		{
			C[cnt] = A[cnt] + B[cnt];
			cnt++;
		}
	}
}

void CheckResult(int *A, int *B, int nx, int ny)
{
	int i, j;
	int cnt = 0;
	for (j = 0; j < ny; j++)
	{
		for (i = 0; i < nx; i++)
		{
			if (abs(A[cnt] - B[cnt]) != 0)
			{
				printf("Do not match...\n");
				return;
			}
		}
	}
	printf("matched...\n");
}

int main(void)
{
	int nx = 1 << 10;
	int ny = 1 << 10;
	int nBytes = sizeof(int)*nx*ny;
	int *h_A, *h_B, *h_C, *gpuRef;
	int *d_A, *d_B, *d_C;
	h_A = (int *)malloc(nBytes);
	h_B = (int *)malloc(nBytes);
	h_C = (int *)malloc(nBytes);
	gpuRef = (int *)malloc(nBytes);

	initalMatrix(h_A, nx, ny);
	initalMatrix(h_B, nx, ny);

	AddMatrixOnCPU(h_A, h_B, h_C, nx, ny);

	hipMalloc((int **)&d_A, nBytes);
	hipMalloc((int **)&d_B, nBytes);
	hipMalloc((int **)&d_C, nBytes);
	
	// memcpy from CPU TO GPU
	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

	dim3 block(32, 1);
	dim3 grid((nx + block.x - 1) / block.x, 1);
	//AddMatrixOnGPU << <grid, block >> >(d_A, d_B, d_C, nx, ny);
	Book << <grid, block >> >(d_A, d_B, d_C, nx, ny);
	hipDeviceSynchronize();

	// copy data from GPU to CPU
	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

	CheckResult(h_C, gpuRef, nx, ny);

	// free the memory
	free(h_A);
	free(h_B);
	free(h_C);
	free(gpuRef);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}
