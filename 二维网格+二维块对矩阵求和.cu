#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

void checkResult(float *A, float *B, const int nx, const int ny)
{
	int i = 0;
	int j = 0;
	int cnt = 0;
	double err = 1.0E-6;
	for (j = 0; j < ny; j++)
	{
		for (i = 0; i < nx; i++)
		{
			if (fabs(A[cnt] - B[cnt]) > err)
			{
				printf("Do not match...\n");
				return;
			}
			cnt++;
		}
	}
	printf("matched!\n");
}

void initialData(float *a, int nx, int ny)
{
	int i = nx;
	int j = ny;
	int cnt = 0;
	for (j = 0; j < ny; j++)
	{
		for (i = 0; i < nx; i++)
		{
			a[cnt] = cnt;
			cnt++;
		}
	}
}

// summary matrix on CPU
void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny)
{
	int i = 0;
	int j = 0;
	int cnt = 0;
	for (j = 0; j < ny; j++)
	{
		for (i = 0; i < nx; i++)
		{
			C[cnt] = A[cnt] + B[cnt];
			cnt++;
		}
	}
}

void PrintMatrix(float *a)
{
	int i;
	for (i = 0; i < 10; i++)
	{
		printf("%f  ", a[i]);
	}
	printf("\n");
}

// summary matrix on GPU
__global__ void sumMatrixOnGPU(float *A, float *B, float *C, int nx, int ny)
{
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = y*nx + x;
	if (x < nx && y < ny)
	{
		C[idx] = A[idx] + B[idx];
	}
}


__global__ void test()
{
	printf("hello\n");
}

int main(int argc, char *argv[])
{
	int dev = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	hipSetDevice(dev);
	int nx = 1 << 10;    // �˴����Կ������ƱȽ����ԣ����п�����nx��ny�ֱ�Ϊ1<<14������nx*ny = 1<<28�������ҵ��Կ����С�
	int ny = 1 << 10;
	int nxy = nx * ny;
	int nBytes = sizeof(float)*nxy;
	printf("Matrix size: nx:%d, ny:%d\n", nx, ny);

	float *h_A, *h_B, *h_C, *gpuRef;
	float *d_A, *d_B, *d_C;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	h_C = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);
	memset(gpuRef, 0, nBytes);

	hipMalloc((void **)&d_A, nBytes);
	hipMalloc((void **)&d_B, nBytes);
	hipMalloc((void **)&d_C, nBytes);

	// initialize the data
	initialData(h_A, nx, ny);
	initialData(h_B, nx, ny);

	// copy the data from CPU to GPU
	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

	// call the summary function
	sumMatrixOnHost(h_A, h_B, h_C, nx, ny);

	dim3 block(32, 32);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
	sumMatrixOnGPU << <grid, block >> >(d_A, d_B, d_C, nx, ny);
	hipDeviceSynchronize();

	// copy the data from GPU to CPU
	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

	// check the result
	checkResult(h_C, gpuRef, nx, ny);

	PrintMatrix(h_C);
	PrintMatrix(gpuRef);

	// free the memory
	free(h_A);
	free(h_B);
	free(h_C);
	free(gpuRef);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipDeviceReset();
	return 0;
}