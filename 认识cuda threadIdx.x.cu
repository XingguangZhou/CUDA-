#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void helloFromGPU(void)
{
	if (threadIdx.x == 5)
	{
		printf("hello from GPU%d!\n", threadIdx.x);
	}
}


int main(void)
{
	printf("Hello world from CPU!\n");
	helloFromGPU << <1, 10 >> > ();
	//hipDeviceReset();
	hipDeviceSynchronize();
	return 0;
}