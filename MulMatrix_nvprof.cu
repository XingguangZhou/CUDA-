#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <windows.h>  // use the QPC 

void GenerateMatrix(float *matrix, int nx, int ny)
{
	int i, j;
	float cnt = 0;
	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			matrix[i*nx + j] = cnt++;
		}
	}
	printf("[*] GenerateMatrix has done!\n");
}

void PrintMatrix(float *matrix, int nx, int ny)
{
	int i, j;
	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			printf("%.2f\t", matrix[i*nx + j]);
		}
		printf("\n");
	}
	printf("[*] PrintMatrix has done!\n");
}

/************************* matrix summary begin *************************/
inline void AddMatrixOnCPU(float *A, float *B, float *C, int nx, int ny)
{
	int i, j;
	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			C[i*nx + j] = A[i*nx + j] + B[i*nx + j];
		}
	}
	printf("[*] AddMatrix on CPU has done!\n");
}

__global__ inline void AddMatrixOnGPU(float *A, float *B, float *C, int nx, int ny)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = i*nx + j;
	if (i <= nx && j <= ny)
	{
		C[idx] = A[idx] + B[idx];
	}
}
/************************* matrix summary done **************************/
//
//
//
/************************ matrix multiply begin *************************/
inline void MulMatrixOnCPU(float *A, float *B, float *C, int nx, int ny)
{
	int i, j, k;
	float sum = 0.0;
	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			sum = 0.0;
			for (k = 0; k < nx; k++)
			{
				sum = sum + A[i*nx + k] * B[k*nx + j];
			}
			C[i*nx + j] = sum;
		}
	}
}

__global__ inline void MulMatrixOnGPU(float *A, float *B, float *C, int nx, int ny)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k;
	if (i < nx && j < ny)   // we should to identify the "i" and "j" scope.
	{
		float sum = 0.0;
		for (k = 0; k < nx; k++)
		{
			sum += A[i*nx + k] * B[k*nx + j];
		}
		C[i*nx + j] = sum;
	}
}
/************************ matrix multiply end ***************************/

// compare the result
int Compare(float *cpu_ref, float *gpu_ref, int nx, int ny)
{
	int i, j;
	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			if (cpu_ref[i*nx + j] != gpu_ref[i*nx + j])
			{
				return 0;
			}
		}
	}
	return 1;
}


int main(int argc, char *argv[])
{
	LARGE_INTEGER begin_cpu, begin_gpu;
	LARGE_INTEGER end_cpu, end_gpu;
	LARGE_INTEGER freq_cpu, freq_gpu;

	// the size of the elements in the matrix can not be much larger....
	// because of my worse GPU: nVIDIA GeForce GT710
	unsigned int N = 1 << 12;
	int nx = (int)sqrt((float)N);
	int ny = (int)sqrt((float)N);

	float *A = NULL;
	float *B = NULL;
	float *C = NULL;
	float *gpu_ref = NULL;
	float *d_A = NULL;
	float *d_B = NULL;
	float *d_C = NULL;

	// allocate the memory on CPU
	A = (float *)malloc(sizeof(float)* N);
	B = (float *)malloc(sizeof(float)* N);
	C = (float *)malloc(sizeof(float)* N);
	gpu_ref = (float *)malloc(sizeof(float)*N);
	// set the memory to zero
	memset(A, 0, sizeof(float)*N);
	memset(B, 0, sizeof(float)*N);
	memset(C, 0, sizeof(float)*N);
	memset(gpu_ref, 0, sizeof(float)*N);

	// allocate the memory on GPU
	hipMalloc((float **)&d_A, sizeof(float)*N);
	hipMalloc((float **)&d_B, sizeof(float)*N);
	hipMalloc((float **)&d_C, sizeof(float)*N);
	// reset the memory to zero
	hipMemset(d_A, 0, sizeof(float)*N);
	hipMemset(d_B, 0, sizeof(float)*N);
	hipMemset(d_C, 0, sizeof(float)*N);

	// generate the matrix on CPU
	GenerateMatrix(A, nx, ny);
	GenerateMatrix(B, nx, ny);

	// transfer the data from CPU to GPU
	hipMemcpy(d_A, A, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(float)*N, hipMemcpyHostToDevice);


	// set the grid number and the block thread number
	dim3 block(32, 32);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

	// Add the matrix on CPU
	AddMatrixOnCPU(A, B, C, nx, ny);

	// Add the matrix on GPU
	AddMatrixOnGPU << <grid, block >> >(d_A, d_B, d_C, nx, ny);
	hipDeviceSynchronize();  // let the CPU wait the GPU to do its calculation.

	// transform the data from the GPU to CPU
	hipMemcpy(gpu_ref, d_C, sizeof(float)*N, hipMemcpyDeviceToHost);

	if (Compare(C, gpu_ref, nx, ny))
	{
		printf("[*] Compare : Matrix_ADD => the result are the same!\n");
	}
	else
	{
		printf("[*] Compare : Matrix_ADD => the result are NOT the same...\n");
	}

	// begin to calculate the time consumption
	QueryPerformanceCounter(&freq_cpu);
	QueryPerformanceCounter(&begin_cpu);

	// test the matrix multiply
	MulMatrixOnCPU(A, B, C, nx, ny);

	QueryPerformanceCounter(&end_cpu);
	printf("CPU time consumption:%f ms\n", 1000*(float)(end_cpu.QuadPart - begin_cpu.QuadPart) / (float)freq_cpu.QuadPart);

	// test the matrix multiply on GPU
	MulMatrixOnGPU << <grid, block >> >(d_A, d_B, d_C, nx, ny);
	hipDeviceSynchronize();

	hipMemcpy(gpu_ref, d_C, sizeof(float)*N, hipMemcpyDeviceToHost);

	// make the comparison
	if (Compare(C, gpu_ref, nx, ny))
	{
		printf("[*] Compare : Matrix_MUL => the result are the same!\n");
	}
	else
	{
		printf("[*] Compare : Matrix_MUL => the result are NOT the same...\n");
	}

	// Debug Print
	// PrintMatrix(gpu_ref, nx, ny);
	// PrintMatrix(C, nx, ny);

	free(A);
	free(B);
	free(C);
	free(gpu_ref);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}


// [*] GenerateMatrix has done!
// [*] GenerateMatrix has done!
// [*] AddMatrix on CPU has done!
// [*] Compare : Matrix_ADD = > the result are the same!
// [*] Compare : Matrix_MUL = > the result are the same!
// Press any key to continue...


// nvprof check
// C:\Users\HP\Desktop\test\x64\Debug > nvprof test.exe
// 	== 18712 == NVPROF is profiling process 18712, command: test.exe
// 	[*] GenerateMatrix has done!
// 	[*] GenerateMatrix has done!
// 	[*] AddMatrix on CPU has done!
// 	[*] Compare : Matrix_ADD = > the result are the same!
// 	CPU time consumption : 0.000002 ms
// 	GPU time consumption : 0.000002 ms
// 	[*] Compare : Matrix_MUL = > the result are the same!
// 	== 18712 == Profiling application : test.exe
// 	== 18712 == Profiling result :
//   Type  Time(%)      Time     Calls       Avg       Min       Max  Name
// 	  GPU activities : 91.91%  718.66us         1  718.66us  718.66us  718.66us  MulMatrixOnGPU(float*, float*, float*, int, int)
// 	  3.62%  28.285us         1  28.285us  28.285us  28.285us  AddMatrixOnGPU(float*, float*, float*, int, int)
// 	  1.93%  15.071us         3  5.0230us  3.8390us  7.3600us[CUDA memset]
// 	  1.28%  10.047us         2  5.0230us  4.9280us  5.1190us[CUDA memcpy DtoH]
// 	  1.26%  9.8870us         2  4.9430us  4.5760us  5.3110us[CUDA memcpy HtoD]
// 	  API calls : 90.76%  331.25ms         3  110.42ms  2.6000us  331.25ms  hipMalloc
// 	  8.46%  30.874ms         1  30.874ms  30.874ms  30.874ms  hipDevicePrimaryCtxRelease
// 	  0.24%  871.50us         4  217.88us  55.900us  641.20us  hipMemcpy
// 	  0.24%  870.40us         3  290.13us  12.400us  790.50us  hipDeviceSynchronize
// 	  0.17%  616.90us         1  616.90us  616.90us  616.90us  hipModuleUnload
// 	  0.07%  242.00us        97  2.4940us     100ns  127.40us  hipDeviceGetAttribute
// 	  0.04%  149.10us         3  49.700us  6.6000us  122.20us  hipFree
// 	  0.01%  47.200us         2  23.600us  15.100us  32.100us  hipLaunchKernel
// 	  0.01%  22.300us         1  22.300us  22.300us  22.300us  hipDeviceTotalMem
// 	  0.00%  14.100us         3  4.7000us  1.4000us  10.600us  hipMemset
// 	  0.00%  6.8000us         1  6.8000us  6.8000us  6.8000us  hipDeviceGetPCIBusId
// 	  0.00%  2.7000us         3     900ns     200ns  2.3000us  hipGetDeviceCount
// 	  0.00%  1.5000us         2     750ns     100ns  1.4000us  hipDeviceGet
// 	  0.00 % 800ns         1     800ns     800ns     800ns  hipDeviceGetName
// 	  0.00 % 400ns         1     400ns     400ns     400ns  hipDeviceGetUuid
// 	  0.00 % 200ns         1     200ns     200ns     200ns  cuDeviceGetLuid
// 
//   C : \Users\HP\Desktop\test\x64\Debug > cd ..
// 
//   C:\Users\HP\Desktop\test\x64 > cd Release
// 
// C : \Users\HP\Desktop\test\x64\Release > nvprof test.exe
// 	== 18808 == NVPROF is profiling process 18808, command: test.exe
// 	[*] GenerateMatrix has done!
// 	[*] GenerateMatrix has done!
// 	[*] AddMatrix on CPU has done!
// 	[*] Compare : Matrix_ADD = > the result are the same!
// 	CPU time consumption : 0.000000 ms
// 	[*] Compare : Matrix_MUL = > the result are the same!
// 	== 18808 == Profiling application : test.exe
// 	== 18808 == Profiling result :
//   Type  Time(%)      Time     Calls       Avg       Min       Max  Name
// 	  GPU activities : 91.07%  599.83us         1  599.83us  599.83us  599.83us  MulMatrixOnGPU(float*, float*, float*, int, int)
// 	  3.82%  25.150us         1  25.150us  25.150us  25.150us  AddMatrixOnGPU(float*, float*, float*, int, int)
// 	  1.97%  12.991us         3  4.3300us  3.6790us  5.6320us[CUDA memset]
// 	  1.61%  10.624us         2  5.3120us  5.3120us  5.3120us[CUDA memcpy HtoD]
// 	  1.53%  10.079us         2  5.0390us  4.8000us  5.2790us[CUDA memcpy DtoH]
// 	  API calls : 73.36%  96.757ms         3  32.252ms  3.1000us  96.746ms  hipMalloc
// 	  25.46%  33.576ms         1  33.576ms  33.576ms  33.576ms  hipDevicePrimaryCtxRelease
// 	  0.52%  691.50us         2  345.75us  59.600us  631.90us  hipDeviceSynchronize
// 	  0.17%  224.60us         4  56.150us  25.500us  81.700us  hipMemcpy
// 	  0.16%  213.70us         1  213.70us  213.70us  213.70us  hipModuleUnload
// 	  0.13%  175.10us         3  58.366us  6.4000us  152.30us  hipFree
// 	  0.12%  157.10us        97  1.6190us     100ns  69.500us  hipDeviceGetAttribute
// 	  0.03%  42.400us         2  21.200us  13.300us  29.100us  hipLaunchKernel
// 	  0.02%  24.400us         1  24.400us  24.400us  24.400us  hipDeviceTotalMem
// 	  0.01%  15.300us         3  5.1000us  1.5000us  11.900us  hipMemset
// 	  0.00%  6.5000us         1  6.5000us  6.5000us  6.5000us  hipDeviceGetPCIBusId
// 	  0.00%  2.6000us         3     866ns     200ns  2.2000us  cuDeviceGetCountt
// 	  0.00%  1.4000us         2     700ns     100ns  1.3000us  hipDeviceGet
// 	  0.00%  1.4000us         1  1.4000us  1.4000us  1.4000us  hipDeviceGetName
// 	  0.00 % 400ns         1     400ns     400ns     400ns  cuDeviceGetLuid
// 	  0.00 % 300ns         1     300ns     300ns     300ns  hipDeviceGetUuid
