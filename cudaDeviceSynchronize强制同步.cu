#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

__global__ void AddArray(int *a, int *b, int *c)
{
	int i = threadIdx.x;
	printf("GPU!\n");
	c[i] = a[i] + b[i];
}


int main(void)
{
	int h_a[4] = { 0, 0, 0, 1 };
	int h_b[4] = { 1, 2, 3, 4 };
	int h_c[4] = { 0 };
	int *d_a = NULL;
	int *d_b = NULL;
	int *d_c = NULL;
	int i;

	dim3 block(4);
	dim3 grid((4 + block.x - 1) / block.x);

	hipMalloc((int **)&d_a, sizeof(int)* 4);
	hipMalloc((int **)&d_b, sizeof(int)* 4);
	hipMalloc((int **)&d_c, sizeof(int)* 4);

	hipMemcpy(d_a, h_a, sizeof(int)* 4, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int)* 4, hipMemcpyHostToDevice);

	AddArray << <grid, block >> >(d_a, d_b, d_c);
	//hipDeviceSynchronize(); // ǿ���豸�������໥ͬ����Ҫ��˺���������ɺ�CPU�ٽ��м��㣬�������飬����������ȥ����鿴�����
	for (i = 0; i < 3; i++)
	{
		printf("CPU!\n");
	}

	hipMemcpy(h_c, d_c, sizeof(int)* 4, hipMemcpyDeviceToHost);

	for (i = 0; i < 4; i++)
	{
		printf("%d\t", h_c[i]);
	}
	printf("\n");

	// free the memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipDeviceReset();
	return 0;
}