#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

void GenerateMatrix(float *matrix, int nx, int ny)
{
	int i, j;
	float cnt = 0;
	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			matrix[i*nx + j] = cnt++;
		}
	}
	printf("[*] GenerateMatrix has done!\n");
}

void PrintMatrix(float *matrix, int nx, int ny)
{
	int i, j;
	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			printf("%.2f\t", matrix[i*nx + j]);
		}
		printf("\n");
	}
	printf("[*] PrintMatrix has done!\n");
}

/************************* matrix summary begin *************************/ 
void AddMatrixOnCPU(float *A, float *B, float *C, int nx, int ny)
{
	int i, j;
	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			C[i*nx + j] = A[i*nx + j] + B[i*nx + j];
		}
	}
	printf("[*] AddMatrix on CPU has done!\n");
}

__global__ void AddMatrixOnGPU(float *A, float *B, float *C, int nx, int ny)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = i*nx + j;
	if (i <= nx && j <= ny)
	{
		C[idx] = A[idx] + B[idx];
	}
}
/************************* matrix summary done **************************/
//
//
//
/************************ matrix multiply begin *************************/
void MulMatrixOnCPU(float *A, float *B, float *C, int nx, int ny)
{
	int i, j, k;
	float sum = 0.0;
	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			sum = 0.0;
			for (k = 0; k < nx; k++)
			{
				sum = sum + A[i*nx + k] * B[k*nx + j];
			}
			C[i*nx + j] = sum;
		}
	}
}

__global__ void MulMatrixOnGPU(float *A, float *B, float *C, int nx, int ny)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k;
	if (i < nx && j < ny)   // we should to identify the "i" and "j" scope.
	{
		float sum = 0.0;
		for (k = 0; k < nx; k++)
		{
			sum += A[i*nx + k] * B[k*nx + j];
		}
		C[i*nx + j] = sum;
	}
}
/************************ matrix multiply end ***************************/

// compare the result
int Compare(float *cpu_ref, float *gpu_ref, int nx, int ny)
{
	int i, j;
	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			if (cpu_ref[i*nx + j] != gpu_ref[i*nx + j])
			{
				return 0;
			}
		}
	}
	return 1;
}


int main(int argc, char *argv[])
{
	LARGE_INTEGER begin_cpu, begin_gpu;
	LARGE_INTEGER end_cpu, end_gpu;
	LARGE_INTEGER freq_cpu, freq_gpu;
	
	// the size of the elements in the matrix can not be much larger....
	// because of my worse GPU: nVIDIA GeForce GT710
	unsigned int N = 1<<12; 
	int nx = (int)sqrt((float)N);
	int ny = (int)sqrt((float)N);

	float *A = NULL;
	float *B = NULL;
	float *C = NULL;
	float *gpu_ref = NULL;
	float *d_A = NULL;
	float *d_B = NULL;
	float *d_C = NULL;

	// allocate the memory on CPU
	A = (float *)malloc(sizeof(float)* N);
	B = (float *)malloc(sizeof(float)* N);
	C = (float *)malloc(sizeof(float)* N);
	gpu_ref = (float *)malloc(sizeof(float)*N);
	// set the memory to zero
	memset(A, 0, sizeof(float)*N);
	memset(B, 0, sizeof(float)*N);
	memset(C, 0, sizeof(float)*N);
	memset(gpu_ref, 0, sizeof(float)*N);

	// allocate the memory on GPU
	hipMalloc((float **)&d_A, sizeof(float)*N);
	hipMalloc((float **)&d_B, sizeof(float)*N);
	hipMalloc((float **)&d_C, sizeof(float)*N);
	// reset the memory to zero
	hipMemset(d_A, 0, sizeof(float)*N);
	hipMemset(d_B, 0, sizeof(float)*N);
	hipMemset(d_C, 0, sizeof(float)*N);

	// generate the matrix on CPU
	GenerateMatrix(A, nx, ny);
	GenerateMatrix(B, nx, ny);

	// transfer the data from CPU to GPU
	hipMemcpy(d_A, A, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(float)*N, hipMemcpyHostToDevice);


	// set the grid number and the block thread number
	dim3 block(32, 32);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

	// Add the matrix on CPU
	AddMatrixOnCPU(A, B, C, nx, ny);

	// Add the matrix on GPU
	AddMatrixOnGPU << <grid, block >> >(d_A, d_B, d_C, nx, ny);
	hipDeviceSynchronize();  // let the CPU wait the GPU to do its calculation.

	// transform the data from the GPU to CPU
	hipMemcpy(gpu_ref, d_C, sizeof(float)*N, hipMemcpyDeviceToHost);

	if (Compare(C, gpu_ref, nx, ny))
	{
		printf("[*] Compare : Matrix_ADD => the result are the same!\n");
	}
	else
	{
		printf("[*] Compare : Matrix_ADD => the result are NOT the same...\n");
	}

	// begin to calculate the time consumption
	QueryPerformanceCounter(&freq_cpu);
	QueryPerformanceCounter(&begin_cpu);
	
	// test the matrix multiply
	MulMatrixOnCPU(A, B, C, nx, ny);
	// because of the GPU calculation use this function, so we should to make the same situation.
	hipDeviceSynchronize();

	QueryPerformanceCounter(&end_cpu);
	printf("CPU time consumption:%f ms\n", 1000 * (float)(end_cpu.QuadPart - begin_cpu.QuadPart) / (float)freq_cpu.QuadPart);

	// begin to calculate the time consumption
	QueryPerformanceCounter(&freq_gpu);
	QueryPerformanceCounter(&begin_gpu);

	// test the matrix multiply on GPU
	MulMatrixOnGPU << <grid, block >> >(d_A, d_B, d_C, nx, ny);
	hipDeviceSynchronize();

	QueryPerformanceCounter(&end_gpu);
	printf("GPU time consumption:%f ms\n", 1000 * (float)(end_gpu.QuadPart - begin_gpu.QuadPart) / (float)freq_gpu.QuadPart);

	hipMemcpy(gpu_ref, d_C, sizeof(float)*N, hipMemcpyDeviceToHost);

	// make the comparison
	if (Compare(C, gpu_ref, nx, ny))
	{
		printf("[*] Compare : Matrix_MUL => the result are the same!\n");
	}
	else
	{
		printf("[*] Compare : Matrix_MUL => the result are NOT the same...\n");
	}

	// Debug Print
	// PrintMatrix(gpu_ref, nx, ny);
	// PrintMatrix(C, nx, ny);
	
	free(A);
	free(B);
	free(C);
	free(gpu_ref);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}


// [*] GenerateMatrix has done!
// [*] GenerateMatrix has done!
// [*] AddMatrix on CPU has done!
// [*] Compare : Matrix_ADD = > the result are the same!
// [*] Compare : Matrix_MUL = > the result are the same!
// Press any key to continue...
